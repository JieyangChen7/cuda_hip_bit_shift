
#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel
__global__ void bit_shift_test()
{
  unsigned long long one = 1;
  for (int i = 0; i <= 64; i++) {
    printf("(%llu << %d) - 1 = %llu\n", one, i, (one << i) - 1);
  }
  unsigned long long max = 18446744073709551615;
  for (int i = 0; i <= 64; i++) {
    printf("%llu >> %d = %llu\n", max, i, max >> i);
  }
}

// Main program
int main()
{
  // Launch kernel
  bit_shift_test<<<1, 1>>>();
  hipError_t cuErrSync  = hipGetLastError();
  hipError_t cuErrAsync = hipDeviceSynchronize();
  if (cuErrSync != hipSuccess) { printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErrSync)); exit(0); }
  if (cuErrAsync != hipSuccess) { printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErrAsync)); exit(0); }

  return 0;
}
